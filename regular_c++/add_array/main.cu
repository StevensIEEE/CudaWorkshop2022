
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

void checkElementsAre(float target, double *vector, int N)
{
    for(int i = 0; i < N; i++)
    {
        if(vector[i] != target)
        {
            printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
            exit(1);
        }
    }
    printf("Success! All values calculated correctly.\n");
}

int main (){

    int n = 100000000;
    double* a = new double[n];
    double* b = new double[n];
    double* c = new double[n];

    for(int i = 0; i< n; i++){
        a[i] = 3;
    }

    for(int i = 0; i< n; i++){
        b[i] = 4;
    }

    for(int i = 0; i<n; i++){
        c[i] = a[i] + b[i];
    }

    checkElementsAre(7,c,n);
    cout << c[75000];
    return 1;
}