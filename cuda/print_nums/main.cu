
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void forloop(){
    printf("%i\n",threadIdx.x) ;
}

int main() {
    //calls for loop with 100 blocks and 1024 threads
    forloop<<<100,1024>>>();
    hipDeviceSynchronize();
    return 0;
}