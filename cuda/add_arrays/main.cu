
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initWith(float num, float *a, int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for(int i = idx; i< N; i+=stride){
        a[i] = num;
    }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < N; i += stride){
        result[i] = a[i] + b[i];
    }
}

void checkElementsAre(float target, float *vector, int N)
{
    for(int i = 0; i < N; i++)
    {
        if(vector[i] != target)
        {
            printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
            exit(1);
        }
    }
    printf("Success! All values calculated correctly.\n");
}

int main(){
    float *a;
    float *b;
    float *c;

    int N = 100000000;
    size_t size = N * sizeof(float);

    int deviceId;
    hipGetDevice(&deviceId);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    size_t threadsPerBlock = 256;
    size_t numberOfBlocks = props.multiProcessorCount *32;

    initWith<<<numberOfBlocks, threadsPerBlock>>>(3,a,N);
    initWith<<<numberOfBlocks, threadsPerBlock>>>(4,b,N);
    initWith<<<numberOfBlocks, threadsPerBlock>>>(0,c,N);
    addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
    hipDeviceSynchronize();
    checkElementsAre(7, c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);
}