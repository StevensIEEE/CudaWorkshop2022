
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void forloop(int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for(int x = idx; x<N; x+=stride){
        printf("%d\n",x);
    }
}

int main() {
    //calls for loop with 100 blocks and 1024 threads
    forloop<<<32,1024>>>(1000000);
    hipDeviceSynchronize();
    return 0;
}