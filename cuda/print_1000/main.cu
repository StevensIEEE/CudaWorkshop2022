#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

__global__ void print(){
    printf("%d\n", threadIdx.x);

}

int main(){
    print<<<1, 500>>>();
    hipDeviceSynchronize();
    return 1;
}

nvcc nameOfFile.cu -o build -run